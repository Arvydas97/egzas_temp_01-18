#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include ""
#include <json.hpp>
#include "Car.h"
using json = nlohmann::json;

using namespace std;
__device__  void filterNewCars(int start_index, int end_index, Car *cars, Car *new_cars, size_t *counter);
__global__ void kernel(Car *cars, Car *new_cars, size_t *counter) ;
void printToFile(Car * result_cars, size_t counter_rez);

const size_t ARRAY_SIZE = 26;
const size_t YEAR_TO_CHECK = 2020;
const float PRICE_TO_CHECK = 15.00;


int main() {
    json cars_json;
    std::ifstream cars_file("data1.json", std::ifstream::binary);
    cars_file >> cars_json;

    Car * cars = new Car[ARRAY_SIZE];
    Car * new_cars = new Car[ARRAY_SIZE];
    Car * result_cars =new Car[ARRAY_SIZE];
    Car * device_cars, *device_new_cars;

    size_t* counter = 0;
    size_t * device_counter;
    size_t counter_rez = 0;

    hipError_t cuda_status;

    size_t i = 0;
    for (auto& to_c : cars_json["autoList"]) {
        Car c = Car(to_c["name"], to_c["year"], to_c["price"]);
        cars[i++]=c; 
    }

    // Pradiniu duomeny perkelimas
    cuda_status = hipMalloc((void**)&device_cars, ARRAY_SIZE*sizeof(Car));
    if(cuda_status != hipSuccess ){
        fprintf(stderr, "hipMalloc error \n");
        exit (0);
    }
    cuda_status = hipMemcpy(device_cars, cars, ARRAY_SIZE*sizeof(Car), hipMemcpyHostToDevice);
    if(cuda_status != hipSuccess ){
        hipFree(device_cars);
        fprintf(stderr, "hipMemcpy error \n");
        exit (0);
    }

    // Atrinktu auto perkelimas
    cuda_status = hipMalloc((void**)&device_new_cars, ARRAY_SIZE*sizeof(Car));
    if(cuda_status != hipSuccess ){
        hipFree(device_cars);
        fprintf(stderr, "hipMalloc error \n");
        exit (0);
    }


    cuda_status = hipMemcpy(device_new_cars, new_cars, ARRAY_SIZE*sizeof(Car), hipMemcpyHostToDevice);
    if(cuda_status != hipSuccess ){
        hipFree(device_cars);
        hipFree(device_new_cars);
        fprintf(stderr, "hipMemcpy error \n");
        exit (0);
    }

    // Counterio perkelimas
    cuda_status = hipMalloc((void**)&device_counter, sizeof(size_t));
    if(cuda_status != hipSuccess ){
        hipFree(device_cars);
        hipFree(device_new_cars);
        fprintf(stderr, "hipMalloc error \n");
        exit (0);
    }
    hipMemcpy(device_counter, counter, sizeof(size_t), hipMemcpyHostToDevice);

    // Kernelis kviecia gpu
    // sinchronizavcija
    kernel <<< 1, 5 >>> (device_cars, device_new_cars, device_counter);
    cuda_status = hipDeviceSynchronize();
    if(cuda_status != hipSuccess ){
        hipFree(device_cars);
        hipFree(device_new_cars);
        hipFree(device_counter);
        fprintf(stderr, "hipDeviceSynchronize error \n");
        exit (0);
    }

    // Rezultatu duomenu perkelimas is gpu
    cuda_status = hipMemcpy(result_cars, device_new_cars, ARRAY_SIZE*sizeof(Car), hipMemcpyDeviceToHost);
    if(cuda_status != hipSuccess ){
        hipFree(device_cars);
        hipFree(device_new_cars);
        hipFree(device_counter);
        fprintf(stderr, "hipMemcpy error \n");
        exit (0);
    }
    cuda_status = hipMemcpy(&counter_rez, device_counter, sizeof(size_t), hipMemcpyDeviceToHost);
    if(cuda_status != hipSuccess ){
        hipFree(device_cars);
        hipFree(device_new_cars);
        hipFree(device_counter);
        fprintf(stderr, "hipMemcpy error \n");
        exit (0);
    }


    printToFile(result_cars, counter_rez);

    hipFree(device_cars);
    hipFree(device_new_cars);
    hipFree(device_counter);
}


void printToFile(Car * result_cars, size_t counter_rez)
{
    ofstream results("results");
    results << "Thread"<<setw(10) << "Code"<<setw(11) << "Price" <<setw(9)<< "Year"<<endl;
    for(size_t i =0; i < counter_rez; i++){
        results<<setw(4) <<result_cars[i].thread <<setw(14) <<result_cars[i].code <<
        setw(9) <<result_cars[i].price<<setw(9) <<result_cars[i].year<< endl;
    }
    results.close();
}

__global__ void kernel(Car *cars, Car *new_cars, size_t *counter) 
{
    int thread_id = threadIdx.x;
    const auto slice_size =ARRAY_SIZE / blockDim.x;
    unsigned int start_index = slice_size*thread_id;
    unsigned int end_index;

    if(thread_id == blockDim.x - 1)
        end_index =ARRAY_SIZE;
    else
        end_index = slice_size*(thread_id + 1);
   
    filterNewCars(start_index, end_index, cars, new_cars, counter);
   
}

__device__ void filterNewCars(int start_index, int end_index, Car *cars, Car *new_cars, size_t *counter)
{
    int thread_id = threadIdx.x;
    size_t value;
    for(auto i = start_index; i < end_index; i++){
        int z = 0;
        char code[20]={}; 
        while (cars[i].name[z] != '\0')
            code[z++] = cars[i].name[z];

        code[z++] = ':';
        code[z++] = ':';

        if(cars[i].price > PRICE_TO_CHECK)
            code[z++] = 'B';
        else
            code[z++] = 'P';
        
        if(YEAR_TO_CHECK ==0){
            value = atomicAdd((unsigned int *)&counter[0], 1);
            for(size_t j =0;j<sizeof(cars[i].code);j++)
                cars[i].code[j] = code[j];
            
            cars[i].thread = thread_id;
            new_cars[value] = cars[i];
        }    
        if( cars[i].year == YEAR_TO_CHECK){
            value = atomicAdd((unsigned int *)&counter[0], 1);
            for(size_t j =0;j<sizeof(cars[i].code);j++)
                cars[i].code[j] = code[j];
            
            cars[i].thread = thread_id;
            new_cars[value] = cars[i];
        }
    }
}